/************************************************
 * MATRIX TRANSPOSE CHECK between parallel
 * 	and sequential programs.
 *
 * Usage:
 * 	Compile using nvcc -lcudart transpose.cu -o transpose
 *	Run using ./mat <size of the matrix>
 *
 * Example:
 *	./mat 153
 *	The above will check whether for a random matrix, A = transpose(A)
 *
 ************************************************/ 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#ifndef __CUDA_SAFE_CALL
hipError_t __cuda_error;
#define __CUDA_SAFE_CALL(call) do { __cuda_error = call; if(__cuda_error != hipSuccess) { fprintf(stderr,"CUDA Error: %s,%s, line %d\n",hipGetErrorString(__cuda_error), __FILE__, __LINE__ ); return -1;} } while(0)
#endif

void generate_random_vector( double *A, int size )
{
	srand( time(NULL) );

	for( int i=0; i<size; i++ )
		//A[i] = 1.2;
		A[i] = ((double)rand())/100000;
}

__device__ void addfour( volatile double *A, int id, int threads, int size, volatile double *B )
{
	int index=threads+id;
	for( int i=0; i<3; i++ ){
		if( index < size ){
			//printf("%d %d %d %d\n",index,id,threads,size);
			B[id] += A[index];
		}
		index += threads;
	}
}

__device__ void maxfour( volatile double *A, int id, int threads, int size, volatile double *B )
{
	int index=threads+id;
	for( int i=0; i<3; i++ ){
		if( index < size ){
			if( B[id] < A[index] )
				//printf("%d %d %d %d\n",index,id,threads,size);
				B[id] = A[index];
		}
		index += threads;
	}
}

__global__ void add4096( double *A, double *B, int size )
{
	__shared__ double nums[256];
	int id = blockIdx.x;
	int tid = threadIdx.x;
	int offset = 4096*id;
	int alive = blockDim.x;
	int this_block = ( size - offset >= 4096 )? 4096: size - offset;
	int intra_offset = 0;
	
	nums[tid] = 0;
	
	while( intra_offset < this_block ){
		if( intra_offset + tid < this_block )
			nums[tid] += A[ offset + intra_offset + tid ];
	       	intra_offset += alive;
	}	

	__syncthreads();
	
	alive = alive >> 2;
	this_block = ( this_block >= blockDim.x )? blockDim.x: this_block;

	while( 1 ){
		if( tid < alive ){
			addfour( (double *)&nums, tid, alive, this_block, (double *)&nums );
			//printf(" id=%d total alive=%d tid=%d, %5.2f\n",id,alive,tid,nums[tid]);
		}
		if( alive == 1 )
			break;
		this_block = ( this_block >= alive )? alive: this_block;
		alive = alive>>2;
		__syncthreads();
	}

	B[id] = nums[0];
}

__global__ void addsquared4096( double *A, double *B, int size )
{
	__shared__ double nums[256];
	int id = blockIdx.x;
	int tid = threadIdx.x;
	int offset = 4096*id;
	int alive = blockDim.x;
	int this_block = ( size - offset >= 4096 )? 4096: size - offset;
	int intra_offset = 0;
	
	nums[tid] = 0;
	
	while( intra_offset < this_block ){
		if( intra_offset + tid < this_block )
			nums[tid] += A[ offset + intra_offset + tid ] * A[ offset + intra_offset + tid ];
	       	intra_offset += alive;
	}	

	__syncthreads();
	
	alive = alive >> 2;
	this_block = ( this_block >= blockDim.x )? blockDim.x: this_block;
	while( 1 ){
		if( tid < alive ){
			addfour( (double *)&nums, tid, alive, this_block, (double *)&nums );
			//printf(" id=%d total alive=%d tid=%d, %5.2f\n",id,alive,tid,nums[tid]);
		}
		if( alive == 1 )
			break;
		this_block = ( this_block >= alive )? alive: this_block;
		alive = alive>>2;
		__syncthreads();
	}

	B[id] = nums[0];
}

__global__ void max4096( double *A, double *B, int size )
{
	__shared__ double nums[256];
	int id = blockIdx.x;
	int tid = threadIdx.x;
	int offset = 4096*id;
	int alive = blockDim.x;
	int this_block = ( size - offset >= 4096 )? 4096: size - offset;
	int intra_offset = 0;
	int tmp;

	nums[tid] = -1;
	
	while( intra_offset < this_block ){
		if( intra_offset + tid < this_block ){
			tmp = A[ offset + intra_offset + tid ];
			if( nums[tid] < tmp )
				nums[tid] = tmp;
		}
	       	intra_offset += alive;
	}	

	__syncthreads();
	
	alive = alive >> 2;
	this_block = ( this_block >= blockDim.x )? blockDim.x: this_block;
	while( 1 ){
		if( tid < alive ){
			maxfour( (double *)&nums, tid, alive, this_block, (double *)&nums );
			//printf(" id=%d total alive=%d tid=%d, %5.2f\n",id,alive,tid,nums[tid]);
		}
		if( alive == 1 )
			break;
		this_block = ( this_block >= alive )? alive: this_block;
		alive = alive>>2;
		__syncthreads();
	}

	B[id] = nums[0];
}

double padd(double *A, int size)
{
	double *ga,*gb;
	int vector_size = sizeof(double) * size;
	int num_blocks = ( ((size - 1) / 4096) + 1 );
	int out_vector = sizeof(double)* num_blocks;
	double ans;

	__CUDA_SAFE_CALL( hipMalloc( &ga, vector_size ) );
	__CUDA_SAFE_CALL( hipMalloc( &gb, out_vector  ) );

	__CUDA_SAFE_CALL( hipMemcpy( ga, A, vector_size, hipMemcpyHostToDevice ) );
	
	while( size > 1 ){
		add4096<<<num_blocks,256>>> (ga, gb, size);
		size = num_blocks;
		num_blocks = ( ((size - 1) / 4096) + 1 );
		ga = gb;
	}

	__CUDA_SAFE_CALL( hipMemcpy( &ans, gb, sizeof(double) , hipMemcpyDeviceToHost ) );
	
	hipFree( ga );
	hipFree( gb );

	return ans;
}

double psquareadd(double *A, int size )
{
	double *ga,*gb;
	int vector_size = sizeof(double) * size;
	int num_blocks = ( ((size - 1) / 4096) + 1 );
	int out_vector = sizeof(double)* num_blocks;
	double ans;

	__CUDA_SAFE_CALL( hipMalloc( &ga, vector_size ) );
	__CUDA_SAFE_CALL( hipMalloc( &gb, out_vector  ) );

	__CUDA_SAFE_CALL( hipMemcpy( ga, A, vector_size, hipMemcpyHostToDevice ) );
	
	while( size > 1 ){
		addsquared4096<<<num_blocks,256>>> (ga, gb, size);
		size = num_blocks;
		num_blocks = ( ((size - 1) / 4096) + 1 );
		ga = gb;
	}

	__CUDA_SAFE_CALL( hipMemcpy( &ans, gb, sizeof(double) , hipMemcpyDeviceToHost ) );
	
	hipFree( ga );
	hipFree( gb );

	return ans;
}

double pmax(double *A, int size )
{
	double *ga,*gb;
	int vector_size = sizeof(double) * size;
	int num_blocks = ( ((size - 1) / 4096) + 1 );
	int out_vector = sizeof(double)* num_blocks;
	double ans;

	__CUDA_SAFE_CALL( hipMalloc( &ga, vector_size ) );
	__CUDA_SAFE_CALL( hipMalloc( &gb, out_vector  ) );

	__CUDA_SAFE_CALL( hipMemcpy( ga, A, vector_size, hipMemcpyHostToDevice ) );
	
	while( size > 1 ){
		max4096<<<num_blocks,256>>> (ga, gb, size);
		size = num_blocks;
		num_blocks = ( ((size - 1) / 4096) + 1 );
		ga = gb;
	}

	__CUDA_SAFE_CALL( hipMemcpy( &ans, gb, sizeof(double) , hipMemcpyDeviceToHost ) );
	
	hipFree( ga );
	hipFree( gb );

	return ans;
}


double pmean( double *A, int size )
{
	double ans;
	
	ans = padd( A, size );
	return ans/size;
}

double pstd( double *A, int size )
{
	double mean, squaredsum;
	mean = pmean( A, size );
	squaredsum = psquareadd( A, size )/size;
	return sqrt(squaredsum-(mean*mean));
}

double sadd( double* A, int size )
{
	double ans=0;
	for( int i=0; i< size; i++ ){
		ans += A[i];
		//printf("%lf \n",ans);
	}
	return ans;
}

double smean( double *A, int size )
{
	double ans;
	ans = sadd( A, size );
	return ans/size;
}

int main( int argc, char* argv[] )
{
	/* Matrix container pointers */
	double *A;

	int size;		/* Number of elements */
	int vector_size;	/* Physical size of the elements in the memory */
	
	hipEvent_t start,stop;
	
	bool do_print=false;	/* Debug flag to print matrices in case of small matrices */
	
	float pms = 0,sms=0;	/* Parallel and sequential times */

	double mean,std,max;
	
	if( argc != 2 ){
		fprintf(stderr, "Atleast one argument required. Usage: %s <Side of the matrix>",argv[0]);
		return -1;
	}
	
	/* Get size of the matrix from command line */
	size = atoi( argv[1] );

	vector_size = sizeof(double)* size;
		
	if( size <= 32 ) do_print= true;

	A = (double *) malloc( vector_size );
	//B = (double *) malloc( out_vector );

	generate_random_vector( A, size );

	if( do_print ){
		for( int i=0; i<size; i++ )
			printf("%5.2f ",A[i]);
	}


	/* Timers to time the parallel process */ 

	__CUDA_SAFE_CALL( hipSetDevice(2) );

	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*********************
	  * Start of GPU run
	  *******************/
	hipEventRecord(start);
	hipEventRecord(stop);
	
	
	mean = 0;
	std = 0;
	max = 0;
	mean = pmean( A, size );
	std = pstd( A, size );
	max = pmax( A, size );

	hipEventSynchronize(stop);
	/*****************
	 * End of GPU code
	 ****************/
	
	hipEventElapsedTime( &pms, start, stop );

	printf("Mean is %lf\n",mean);
	printf("Std is %lf\n",std);
	printf("Max is %lf\n",max);
	/*********************
	 * Sequential Stuff
	 ********************/
	struct timespec seq_start,seq_end;

	/* clock_gettime gets the process specific time spent, as opposed to the system time expended
	 */
	clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &seq_start );
	
	mean = smean( A, size );	

	clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &seq_end );

	/*************************
	 * End of Sequential Stuff
	 ************************/

	printf("Mean is %lf\n",mean);
	
	/* Getting time in milliseconds for comparability */
	sms = ( (float)seq_end.tv_sec - seq_start.tv_sec )*1000 + ( (float)seq_end.tv_nsec - seq_start.tv_nsec ) / 1000000;
	printf("%12s %12s %12s %12s\n","N","Parallel","Sequential","Speedup");
	printf("%12d % 12f % 12f % 12f\n",size,pms,sms,sms/pms);
	/*
	printf("<html>\n\t<body>\n\t\t<table>\n");
	printf("<tr>\n");
	printf("\t<td> %12d </td>\n\t<td>% 12f</td>\n\t<td>% 12f</td>\n\t<td>% 12f</td>\n",size,pms,sms,sms/pms);
	printf("</tr>\n");
	printf("</table>\n</body>\n</html>\n");
	*/
	free(A);
}





